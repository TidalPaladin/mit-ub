#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <torch/extension.h>

#define WARP_SIZE 32

template <typename scalar_t>
__global__ void invert_kernel(const scalar_t *__restrict__ input, scalar_t *__restrict__ output,
                              const float invert_prob, const int64_t batch_size, const int64_t seq_len,
                              const int64_t seed) {
    const int batch_idx = blockIdx.y;
    const int seq_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (seq_idx >= seq_len || batch_idx >= batch_size) return;

    // Load input value
    const int64_t idx = batch_idx * seq_len + seq_idx;
    scalar_t val = input[idx];

    // Decide if this batch entry should be inverted
    hiprandState batch_state;
    hiprand_init(seed + batch_idx, 0, 0, &batch_state);
    const bool apply_invert = hiprand_uniform(&batch_state) < invert_prob;
    if (!apply_invert) {
        output[idx] = val;
        return;
    } else {
        output[idx] = 1.0f - val;
    }
}

torch::Tensor invert(const torch::Tensor &input, const float invert_prob, const int64_t seed) {
    // Prepare output and infer dimensions
    auto output = torch::empty_like(input);
    const int64_t batch_size = input.size(0);
    const int64_t seq_len = input.numel() / batch_size;

    AT_DISPATCH_FLOATING_TYPES_AND2(
        at::ScalarType::Half, at::ScalarType::BFloat16, input.scalar_type(), "invert", ([&] {
            int min_grid_size;
            int block_size;
            hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size, (void *)invert_kernel<scalar_t>, 0, 0);
            const unsigned int blocks_x = (seq_len + block_size - 1) / block_size;
            const dim3 blocks(blocks_x, batch_size);

            invert_kernel<scalar_t><<<blocks, block_size>>>(input.data_ptr<scalar_t>(), output.data_ptr<scalar_t>(),
                                                            invert_prob, batch_size, seq_len, seed);
        }));

    return output;
}

void invert_(torch::Tensor &input, const float invert_prob, const int64_t seed) {
    // Prepare output and infer dimensions
    const int64_t batch_size = input.size(0);
    const int64_t seq_len = input.numel() / batch_size;

    AT_DISPATCH_FLOATING_TYPES_AND2(
        at::ScalarType::Half, at::ScalarType::BFloat16, input.scalar_type(), "invert", ([&] {
            int min_grid_size;
            int block_size;
            hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size, (void *)invert_kernel<scalar_t>, 0, 0);
            const unsigned int blocks_x = (seq_len + block_size - 1) / block_size;
            const dim3 blocks(blocks_x, batch_size);

            invert_kernel<scalar_t><<<blocks, block_size>>>(input.data_ptr<scalar_t>(), input.data_ptr<scalar_t>(),
                                                            invert_prob, batch_size, seq_len, seed);
        }));
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("invert", &invert, "Invert operation");
    m.def("invert_", &invert_, "Invert operation in-place");
}
