#include "hip/hip_runtime.h"
/*
Implements a fused kernel for applying various noise types to a batch of images.
Each noise type is either applied or not applied to each entry of the batch independently.
*/
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <torch/extension.h>

// Initialize hiprand states for each thread
__global__ void setup_noise_seq_curand(hiprandState *states, unsigned long seed) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, id, 0, &states[id]);
}

template <typename scalar_t>
__global__ void fused_noise_kernel(const scalar_t *__restrict__ input, scalar_t *__restrict__ output,
                                   const float uniform_noise_min, const float uniform_noise_max,
                                   const float multiplicative_min, const float multiplicative_max,
                                   const float salt_pepper_min, const float salt_pepper_max, const float uniform_prob,
                                   const float multiplicative_prob, const float salt_pepper_prob, const bool clip,
                                   const int64_t batch_size, const int64_t seq_len, const int64_t seed) {
    const int batch_idx = blockIdx.y;
    const int seq_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (seq_idx >= seq_len || batch_idx >= batch_size) return;

    // Load input value
    const int64_t idx = batch_idx * seq_len + seq_idx;
    scalar_t val = input[idx];

    // Compute masks for each noise type for this batch entry
    hiprandState batch_state;
    hiprand_init(seed + batch_idx, 0, 0, &batch_state);
    const scalar_t uniform_mask = hiprand_uniform(&batch_state) < uniform_prob;
    const scalar_t mult_mask = hiprand_uniform(&batch_state) < multiplicative_prob;
    const scalar_t sp_mask = hiprand_uniform(&batch_state) < salt_pepper_prob;

    // Initialize pointwise-unique random state
    hiprandState seq_state;
    const unsigned long seq_seed = seed + batch_idx * seq_len + seq_idx;
    hiprand_init(seq_seed, 0, 0, &seq_state);

    // Compute multiplicative factor (will be 1.0 if not applied)
    const float mult_center = (multiplicative_min + multiplicative_max) / 2.0f;
    const float mult_min_range = multiplicative_min + (mult_center - multiplicative_min) * hiprand_uniform(&seq_state);
    const float mult_max_range = mult_center + (multiplicative_max - mult_center) * hiprand_uniform(&seq_state);
    const float mult_noise = mult_min_range + (mult_max_range - mult_min_range) * hiprand_uniform(&seq_state);
    const float mult_factor = __fmaf_rn(mult_noise - 1.0f, mult_mask, 1.0f);

    // Compute additive factor (will be 0.0 if not applied)
    const float unif_center = (uniform_noise_min + uniform_noise_max) / 2.0f;
    const float unif_min_range = uniform_noise_min + (unif_center - uniform_noise_min) * hiprand_uniform(&seq_state);
    const float unif_max_range = unif_center + (uniform_noise_max - unif_center) * hiprand_uniform(&seq_state);
    const float unif_noise = unif_min_range + (unif_max_range - unif_min_range) * hiprand_uniform(&seq_state);
    const float add_factor = unif_noise * uniform_mask;

    // Compute salt & pepper value (will not be used if not applied)
    const float sp_prob = salt_pepper_min + (salt_pepper_max - salt_pepper_min) * hiprand_uniform(&seq_state);
    const float sp_trigger = hiprand_uniform(&seq_state) < sp_prob;
    const float sp_value = hiprand_uniform(&seq_state) < 0.5f ? 0.0f : 1.0f;

    // Fused multiply-add for noise application
    val = __fmaf_rn(val, mult_factor, add_factor);

    // Blend with salt & pepper value if applied
    const float sp_blend = sp_mask * sp_trigger;
    val = val * (1.0f - sp_blend) + sp_value * sp_blend;

    // Clip using min/max intrinsics if requested
    if (clip) {
        val = __saturatef(val);
    }

    output[idx] = val;
}

torch::Tensor fused_noise_cuda(const torch::Tensor &input, const float uniform_noise_min, const float uniform_noise_max,
                               const float multiplicative_min, const float multiplicative_max,
                               const float salt_pepper_min, const float salt_pepper_max, const float uniform_prob,
                               const float multiplicative_prob, const float salt_pepper_prob, const bool clip,
                               const int64_t seed, const bool inplace) {
    // Prepare output and infer dimensions
    auto output = inplace ? input : torch::empty_like(input);
    const int64_t batch_size = input.size(0);
    const int64_t seq_len = input.numel() / batch_size;

    AT_DISPATCH_FLOATING_TYPES(
        input.scalar_type(), "fused_noise_cuda", ([&] {
            // Calculate grid dimensions
            int min_grid_size;
            int block_size;
            hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size, (void *)fused_noise_kernel<scalar_t>, 0, 0);
            const unsigned int blocks_x = (seq_len + block_size - 1) / block_size;
            const dim3 blocks(blocks_x, batch_size);

            fused_noise_kernel<scalar_t><<<blocks, block_size>>>(
                input.data_ptr<scalar_t>(), output.data_ptr<scalar_t>(), uniform_noise_min, uniform_noise_max,
                multiplicative_min, multiplicative_max, salt_pepper_min, salt_pepper_max, uniform_prob,
                multiplicative_prob, salt_pepper_prob, clip, batch_size, seq_len, seed);
        }));

    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) { m.def("fused_noise", &fused_noise_cuda, "Fused noise operations (CUDA)"); }
